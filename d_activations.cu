#include "hip/hip_runtime.h"
#include "d_activations.cuh"
#include "cuda_kernels.cuh"
#include "D_Tensor.cuh"

#include <math.h>
#include <cstring>

D_Tensor* scale(D_Tensor *mat, bool inplace) {
    D_Tensor *result;

    if (inplace) {
        result = mat;
    } else {
        result = new D_Tensor(mat->rows, mat->cols);
        std::memcpy(result->data, mat->data, sizeof(float) * mat->size);
    }

    float scale_val = std::sqrt(result->cols);

    int grid_size = (result->size + result->block_size - 1) / result->block_size;
    d_scale<<<grid_size, result->block_size>>>(result->data, result->size, scale_val);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
    
    return result;
}

D_Tensor* softmax(D_Tensor *mat, bool inplace){
    D_Tensor *result;

    if (inplace){
        result = mat;
    }
    else{
        result = new D_Tensor(mat->rows, mat->cols);
        std::memcpy(result->data, mat->data, sizeof(float) * mat->size);
    }

    int grid_size = (result->rows + result->block_size - 1) / result->block_size;
    _d_softmax<<<grid_size, result->block_size>>>(result->data, result->rows, result->cols);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    return result;
}